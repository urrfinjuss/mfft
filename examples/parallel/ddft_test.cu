
#include "hip/hip_runtime.h"

#include "hip/hip_math_constants.h"

#include"hip/hip_complex.h"
#include <stdlib.h>
#include <stdio.h>


//A wrapper function to call cuda safely (A ton of error checks)
hipError_t cudaDDFT(unsigned nbits, unsigned nthreads, hipDoubleComplex * out, hipDoubleComplex * in);

__global__ void ddftKernel(unsigned nbits, unsigned nthreads, hipDoubleComplex* out, hipDoubleComplex* in, hipDoubleComplex* W)
{
	long int N = 1 << nbits;
	long int Nloc = N / nthreads;
	long int tid = threadIdx.x;
	long int istart = tid*Nloc;
	long int ifinish = (tid+1)*Nloc;
	for (int k = istart; k < ifinish; k++) {
		out[k] = in[N - 1];
		for (int j = 0; j < N - 1; j++) {
			out[k].x = out[k].x * W[k].x - out[k].y * W[k].y + in[N - 2 - j].x;
			out[k].y = out[k].y * W[k].x + out[k].x * W[k].y + in[N - 2 - j].y;
		}
	}
}

int main()
{
	unsigned nthreads = 16;
	unsigned nbits = 10;

	//Create test
	hipDoubleComplex* in = (hipDoubleComplex*)malloc(1 << nbits * sizeof(hipDoubleComplex));
	hipDoubleComplex* out = (hipDoubleComplex*)malloc(1 << nbits * sizeof(hipDoubleComplex));

	FILE *fh = fopen("init.txt", "w");
	for (unsigned i = 0; i < 1 << nbits; ++i)
	{
		in[i].x = sin(2. * i * HIP_PI / (1 << nbits));
		in[i].y = 0;
		fprintf(fh, "%19.12e\t%19.12e\t%19.12e\n", 2.*HIP_PI*i / (1 << nbits), in[i].x, in[i].y);
	}
	fclose(fh);


	//Measure time


    hipError_t cudaStatus = cudaDDFT(nbits, nthreads, out, in);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	fh = fopen("output.txt", "w");
	for (unsigned i = 0; i < 1 << nbits; ++i)
	{
		fprintf(fh, "%19.12e\t%19.12e\t%19.12e\n", 2.*HIP_PI*i / (1 << nbits), out[i].x, out[i].y);
	}
	fclose(fh);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to compute discrete Fourier transform in parallel.
hipError_t cudaDDFT(unsigned nbits, unsigned nthreads, hipDoubleComplex * out, hipDoubleComplex * in)
{
	hipDoubleComplex* dev_in;
	hipDoubleComplex* dev_out;
	hipDoubleComplex* dev_W;
	hipDoubleComplex* W;

	W = (hipDoubleComplex*)malloc((1 << nbits) * sizeof(hipDoubleComplex));
	// (serial n^2) algorithm prepare twiddle factors
	double phs = -2.0*HIP_PI / (1 << nbits);
	for (int s = 0; s < (1 << nbits); s++) 
	{
		W[s].x = cos(-phs*s);
		W[s].y = sin(-phs*s);
	}
	hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for two vectors (one input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_out, (1 << nbits) * sizeof(hipDoubleComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_in, (1 << nbits) * sizeof(hipDoubleComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_W, (1 << nbits) * sizeof(hipDoubleComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_in, in, (1<<nbits) * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_W, W, (1 << nbits) * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    // Launch a kernel on the GPU with one thread for each element.
    ddftKernel<<<1, nthreads>>>(nbits, nthreads, dev_out, dev_in, dev_W);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_out, (1 << nbits) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_out);
    hipFree(dev_in);
    hipFree(dev_W);
    
    return cudaStatus;
}
